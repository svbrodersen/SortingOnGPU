#include "hip/hip_runtime.h"
// bench_cub_like_futhark.cu
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <random>
#include <cmath>
#include <algorithm> 
#include <string>    
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

static void ck(hipError_t e, const char* where){
  if(e != hipSuccess){
    std::fprintf(stderr, "CUDA error at %s: %s\n", where, hipGetErrorString(e));
    std::exit(1);
  }
}

static std::string with_underscores(size_t n){
  // format e.g. 1000000 -> "1_000_000"
  std::string s = std::to_string(n);
  std::string out; out.reserve(s.size() + s.size()/3);
  int cnt = 0;
  for (int i = (int)s.size()-1; i >= 0; --i){
    out.push_back(s[i]);
    cnt++;
    if (cnt == 3 && i != 0){ out.push_back('_'); cnt = 0; }
  }
  std::reverse(out.begin(), out.end());
  return out;
}

int main(int argc, char** argv){
  if (argc < 2){
    std::fprintf(stderr, "Usage: %s <N> [runs]\n", argv[0]);
    return 1;
  }
  const size_t N = std::strtoull(argv[1], nullptr, 10);
  const int runs = (argc >= 3) ? std::atoi(argv[2]) : 10;

  // Host data (deterministic so results are reproducible)
  std::vector<unsigned int> h(N);
  std::mt19937 rng(123456);
  std::uniform_int_distribution<unsigned int> dist(0, 0xffffffffu);
  for (size_t i = 0; i < N; ++i) h[i] = dist(rng);

  // Device buffers: original input, working input, and output
  unsigned int *d_orig=nullptr, *d_in=nullptr, *d_out=nullptr;
  ck(hipMalloc(&d_orig, N*sizeof(unsigned int)), "hipMalloc d_orig");
  ck(hipMalloc(&d_in,   N*sizeof(unsigned int)), "hipMalloc d_in");
  ck(hipMalloc(&d_out,  N*sizeof(unsigned int)), "hipMalloc d_out");
  ck(hipMemcpy(d_orig, h.data(), N*sizeof(unsigned int), hipMemcpyHostToDevice), "HtoD d_orig");

  // CUB temp storage
  void* d_temp = nullptr; size_t temp_bytes = 0;
  hipcub::DeviceRadixSort::SortKeys(d_temp, temp_bytes, d_in, d_out, N, 0, 32);
  ck(hipMalloc(&d_temp, temp_bytes), "hipMalloc temp");

  // Warmup: also ensures kernels are JITed and clocks up
  for (int i = 0; i < 3; ++i){
    ck(hipMemcpy(d_in, d_orig, N*sizeof(unsigned int), hipMemcpyDeviceToDevice), "D2D warmup");
    hipcub::DeviceRadixSort::SortKeys(d_temp, temp_bytes, d_in, d_out, N, 0, 32);
  }
  ck(hipDeviceSynchronize(), "sync warmup");

  // Time per-run with CUDA events so we can produce CI
  std::vector<float> times_ms; times_ms.reserve(runs);
  hipEvent_t start, stop;
  ck(hipEventCreate(&start), "event start");
  ck(hipEventCreate(&stop),  "event stop");

  for (int r = 0; r < runs; ++r){
    ck(hipMemcpy(d_in, d_orig, N*sizeof(unsigned int), hipMemcpyDeviceToDevice), "D2D reset");
    ck(hipEventRecord(start), "record start");
    hipcub::DeviceRadixSort::SortKeys(d_temp, temp_bytes, d_in, d_out, N, 0, 32);
    ck(hipEventRecord(stop),  "record stop");
    ck(hipEventSynchronize(stop), "sync stop");
    float ms = 0.0f;
    ck(hipEventElapsedTime(&ms, start, stop), "elapsed");
    times_ms.push_back(ms);
  }

  ck(hipEventDestroy(start), "destroy start");
  ck(hipEventDestroy(stop),  "destroy stop");

  // Stats: mean and 95% CI of the mean
  double sum = 0.0;
  for (float v : times_ms) sum += v;
  double mean_ms = sum / runs;

  double var = 0.0;
  for (float v : times_ms){
    double d = v - mean_ms;
    var += d * d;
  }
  var /= (runs > 1 ? (runs - 1) : 1);
  double sd_ms = std::sqrt(var);
  double se_ms = sd_ms / std::sqrt((double)runs);
  // 95% CI using normal approx (for n=10 it’s fine): mean ± 1.96 * SE
  double half95_ms = 1.96 * se_ms;

  // Units like Futhark: microseconds (μs)
  double mean_us   = mean_ms * 1000.0;
  double lo95_us   = (mean_ms - half95_ms) * 1000.0;
  double hi95_us   = (mean_ms + half95_ms) * 1000.0;

  // Throughput (Gkeys/s) for convenience
  double gkeys = (double)N / 1e6 / mean_ms;

  // Print in the same style as futhark bench lines:
  // n=1_000:        383μs (95% CI: [     381.8,      383.9])
  auto label = with_underscores(N);
  std::printf("n=%s: %11.0fμs (95%% CI: [%11.1f, %11.1f])  // %.6f Gkeys/s\n",
              label.c_str(), mean_us, lo95_us, hi95_us, gkeys);

  // Cleanup
  hipFree(d_temp);
  hipFree(d_out);
  hipFree(d_in);
  hipFree(d_orig);
  return 0;
}
