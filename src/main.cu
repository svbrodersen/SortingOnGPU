#include "constants.cuh"
#include "host_skel.cuh"
#include "kernels.cuh"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void printArray(uint32_t *inp_vals, uint32_t N, const char *name) {
  printf("%s[:%d] = [", name, N);
  for (int i = 0; i < N; i++) {
    if (i == N - 1) {
      printf("%u]\n", inp_vals[i]);
    } else {
      printf("%u, ", inp_vals[i]);
    }
  }
}

void printDeviceArray(uint32_t *inp_vals, int mem_size, uint32_t N,
                      const char *name) {
  uint32_t *d_hist_host = (uint32_t *)malloc(mem_size);
  hipMemcpy(d_hist_host, inp_vals, mem_size, hipMemcpyDeviceToHost);
  printArray(d_hist_host, N, name);
}

int radixSort(uint32_t *inp_vals, uint32_t *out_vals, uint32_t N) {
  const uint32_t Q = 22;
  const uint32_t B = 256;
  const uint32_t lgH = 8;
  const uint32_t H = (1 << lgH);
  const uint32_t T = 32;

  const uint32_t mem_size = N * sizeof(uint32_t);

  uint32_t *d_inp_vals;
  uint32_t *d_out_vals;
  hipMalloc((void **)&d_inp_vals, mem_size);
  hipMemcpy(d_inp_vals, inp_vals, mem_size, hipMemcpyHostToDevice);

  // Allocate device output
  hipMalloc((void **)&d_out_vals, mem_size);

  const uint32_t num_blocks = (N + (B * Q) - 1) / (B * Q);
  const uint32_t hist_size = num_blocks * H;
  const uint32_t hist_mem_size = hist_size * sizeof(uint32_t);

  uint32_t current_shift = 0;

  uint32_t *d_hist;
  hipMalloc((void **)&d_hist, hist_mem_size);

  initial_kernel<H, lgH, Q>
      <<<num_blocks, B>>>(d_inp_vals, d_hist, current_shift, N);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();
  printf("Successfully initial_kernel.\n");
  printDeviceArray(d_hist, hist_mem_size, 256, "d_hist");

  uint32_t *d_hist_tr;
  hipMalloc((void **)&d_hist_tr, hist_mem_size);

  uint32_t *d_hist_scan;
  hipMalloc((void **)&d_hist_scan, hist_mem_size);

  int dimy = (num_blocks + T - 1) / T;
  int dimx = (H + T - 1) / T;
  dim3 block(T, T, 1);
  dim3 grid(dimx, dimy, 1);

  transpose<T><<<grid, block>>>(d_hist, d_hist_tr, num_blocks, H);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();

  printf("Successfully transpose.\n");
  printDeviceArray(d_hist_tr, hist_mem_size, 256, "d_hist_tr");

  // Allocate temporary arrays for scanInc
  uint32_t *d_tmp_vals;
  hipMalloc((void **)&d_tmp_vals, num_blocks * sizeof(uint32_t));

  scanInc<Add<uint32_t>>(B, hist_size, d_hist_tr, d_hist_tr, d_tmp_vals);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();

  printf("Successfully scanInc.\n");

  transpose<T><<<grid, block>>>(d_hist_tr, d_hist_scan, num_blocks, H);
  printf("Successfully transpose.\n");

  printDeviceArray(d_hist_tr, hist_mem_size, H, "d_hist_scan");

  const uint32_t shared_mem_size = (2 * B * Q + 2 * H + B) * sizeof(uint32_t);

  final_kernel<H, lgH, B, Q><<<num_blocks, B, shared_mem_size>>>(
      d_inp_vals, d_out_vals, d_hist, d_hist_scan, current_shift, N);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();

  printf("Successfully final_kernel.\n");

  // Copy result back to host (assuming this is the only pass for demonstration)
  hipMemcpy(out_vals, d_out_vals, mem_size, hipMemcpyDeviceToHost);

  hipFree(d_inp_vals);
  hipFree(d_out_vals);
  hipFree(d_hist);
  hipFree(d_hist_tr);
  hipFree(d_tmp_vals);

  return 0;
}

int main() {
  initHwd();

  const uint32_t N = 1u << 20;
  const uint32_t mem_size = N * sizeof(uint32_t);

  uint32_t *inp_vals = (uint32_t *)malloc(mem_size);
  for (int i = 0; i < N; i++) {
    inp_vals[i] = rand();
  }

  printArray(inp_vals, 10, "inp_vals");

  uint32_t *out_vals = (uint32_t *)malloc(mem_size);

  if (radixSort(inp_vals, out_vals, N) == 0) {
    printArray(out_vals, 100, "out_vals");

    // Simple verification for the first pass (lowest 8 bits)
    bool sorted = true;
    for (uint32_t i = 0; i < N - 1; i++) {
      if ((out_vals[i] & 0xFF) > (out_vals[i + 1] & 0xFF)) {
        sorted = false;
        printf("Sort failed at index %u: %u (bin %u) > %u (bin %u)\n", i,
               out_vals[i], out_vals[i] & 0xFF, out_vals[i + 1],
               out_vals[i + 1] & 0xFF);
        break;
      }
    }
    if (sorted) {
      printf("Array is correctly sorted by the lowest 8 bits.\n");
    }
  }

  free(inp_vals);
  free(out_vals);

  return 0;
}
