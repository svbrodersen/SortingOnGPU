#include "constants.cuh"
#include "definitions.cuh"
#include "host_skel.cuh"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
  initHwd();

  const uint32_t array_length = 1 << 20;
  const uint32_t mem_size = array_length * sizeof(uint32_t);

  uint32_t *inp_vals = (uint32_t *)malloc(mem_size);
  for (int i = 0; i < array_length; i++) {
    inp_vals[i] = rand();
  }

  uint32_t *d_inp_vals;
  hipMalloc((void **)&d_inp_vals, mem_size);
  hipMemcpy(d_inp_vals, inp_vals, mem_size, hipMemcpyHostToDevice);

  const uint32_t num_blocks = (array_length + (B * Q) - 1) / (B * Q);
  const uint32_t hist_size = num_blocks * H * sizeof(uint32_t);
  uint32_t *d_hist;
  hipMalloc((void **)&d_hist, hist_size);

  initial_kernel<<<num_blocks, B>>>(d_inp_vals, d_hist, 0);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();
  printf("Successfully initial_kernel.\n");

  uint32_t *d_hist_tr;
  hipMalloc((void **)&d_hist_tr, hist_size);

  int n = num_blocks;
  int m = H;

  int dimy = (n + TILE_DIM - 1) / TILE_DIM;
  int dimx = (m + TILE_DIM - 1) / TILE_DIM;
  dim3 block(TILE_DIM, TILE_DIM, 1);
  dim3 grid(dimx, dimy, 1);

  transpose<<<grid, block>>>(d_hist, d_hist_tr, n, m);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();

  printf("Successfully transpose.\n");

  // Allocate temporary arrays for scanInc
  uint32_t *d_tmp_vals;
  hipMalloc((void **)&d_tmp_vals, num_blocks * sizeof(uint32_t));

  // Run scan on each row of transposed histogram
  for (int i = 0; i < H; i++) {
    scanInc<Add<uint32_t>>(B, num_blocks, d_hist_tr + i * num_blocks,
                           d_hist_tr + i * num_blocks, d_tmp_vals);
  }
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();

  printf("Successfully scanInc.\n");

  // Transpose result back to original layout
  transpose<<<grid, block>>>(d_hist_tr, d_hist, m, n);
  CUDASSERT(hipPeekAtLastError());
  hipDeviceSynchronize();

  printf("Successfully scanInc.\n");

  free(inp_vals);
  hipFree(d_inp_vals);
  hipFree(d_hist);
  hipFree(d_hist_tr);
  hipFree(d_tmp_vals);

  return 0;
}
